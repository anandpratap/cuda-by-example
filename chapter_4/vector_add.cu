#include "hip/hip_runtime.h"
#include "../common/book.h"
#include "assert.h"
#define N 65000

__global__ void add(int *a, int *b, int *c){
  int tid = blockIdx.x;
  if(tid < N){
    c[tid] = a[tid] + b[tid];
  }
}

int main(void){
  int a[N], b[N], c[N];
  int *dev_a, *dev_b, *dev_c;
  // allocate device memory
  hipMalloc((void **)&dev_a, N*sizeof(int));
  hipMalloc((void **)&dev_b, N*sizeof(int));
  hipMalloc((void **)&dev_c, N*sizeof(int));
  
  // init array
  for(int i=0; i < N; i++){
    a[i] = -i;
    b[i] = i*i;
  }

  hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);


  add<<<N,1>>>(dev_a, dev_b, dev_c);

  hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);

  // check the results
  for(int i=0; i < N; i++){
    assert(c[i] == a[i] + b[i]);
  }

  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);
  return 0;
}
