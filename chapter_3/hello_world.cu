#include "hip/hip_runtime.h"
#include "../common/book.h"

__global__ void add(int a, int b, int *c){
  *c = a + b;
}

int main(void){

  // add two numbers a and b and save the results in c
  int c;
  int *dev_c;
  HANDLE_ERROR(hipMalloc((void **)&dev_c, sizeof(int)));
  add<<<1,1>>>(2, 7, dev_c);
  HANDLE_ERROR(hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost));
  printf("2 + 7 = %d\n", c);
  hipFree(dev_c);

  // get device properties
  hipDeviceProp_t prop;
  int count;
  HANDLE_ERROR(hipGetDeviceCount(&count));
  for(int i=0; i<count;i++){
    HANDLE_ERROR(hipGetDeviceProperties(&prop, i));
    printf("Device name: %s\n", prop.name);
    printf("Total global memory: %lu bytes\n", prop.totalGlobalMem);
    printf("Shared memory per block: %lu bytes\n", prop.sharedMemPerBlock);
    printf("Multiprocessor count: %d\n", prop.multiProcessorCount);
    printf("Max thread per block: %d\n", prop.maxThreadsPerBlock);
    printf("Max thread dimensions: (%d, %d, %d)\n", prop.maxThreadsDim[0],
	   prop.maxThreadsDim[1],
	   prop.maxThreadsDim[2]);
    printf("Max grid dimensions: (%d, %d, %d)\n", prop.maxGridSize[0],
	   prop.maxGridSize[1],
	   prop.maxGridSize[2]);
    printf("compute capability %d.%d\n", prop.major, prop.minor);
  }
  return 0;
}
